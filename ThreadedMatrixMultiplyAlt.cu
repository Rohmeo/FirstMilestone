#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

typedef struct{
	int width;
	int height;
	int* elements;
} Matrix;

__global__ void matrixProduct(Matrix, Matrix, Matrix);

main()
{
	//Declare vars, constants
	int const MatSize=9;
	Matrix Matrix1, Matrix2, Matrix3, Result;
	Matrix dev_Matrix1, dev_Matrix2, dev_Result;
	
	Matrix1.width = MatSize; Matrix1.height = MatSize;
	Matrix2.width = MatSize; Matrix2.height = MatSize;
	Result.width = MatSize; Result.height = MatSize;
	Matrix3.width = MatSize; Matrix3.height = MatSize;
	 
	dim3 blockSize(MatSize+1,MatSize+1);
	dim3 gridSize(1,1);

	int i,j;
	struct timeval start, postmem, end;

	size_t MemSize = (MatSize+1) * (MatSize+1) * sizeof(int);
	Matrix1.elements = (int*) malloc(MemSize);
	Matrix2.elements = (int*) malloc(MemSize);
	Matrix3.elements = (int*) malloc(MemSize);
	Result.elements = (int*) malloc(MemSize);
	
	//Initialize matrices with random values
	for(i=0;i<=MatSize;i++)
	{
		for(j=0;j<=MatSize;j++)
		{
			Matrix1.elements[i*Matrix1.width+j]=i*Matrix1.width+j;
			Matrix2.elements[i*Matrix1.width+j]=i*Matrix1.width+j;
		}
	}
	gettimeofday(&start,NULL);
	printf("Start Values %ld, %ld\n",start.tv_sec,start.tv_usec);
	
	//Transfer matrices to device memory
	
	dev_Matrix1.height = Matrix1.height; dev_Matrix1.width = Matrix1.width;
	hipMalloc((void**)&dev_Matrix1.elements,MemSize);
	hipMemcpy(dev_Matrix1.elements, Matrix1.elements, MemSize, hipMemcpyHostToDevice);

	dev_Matrix2.height = Matrix2.height; dev_Matrix2.width = Matrix2.width;
	hipMalloc((void**)&dev_Matrix2.elements,MemSize);
	hipMemcpy(dev_Matrix2.elements, Matrix2.elements, MemSize, hipMemcpyHostToDevice);

	hipMemcpy(Matrix3.elements, dev_Matrix2.elements, MemSize, hipMemcpyDeviceToHost);
	
	dev_Result.height = Result.height; dev_Result.width = Result.width;
	hipMalloc((void**)&dev_Result.elements,MemSize);
	
		
	//Kernel Declaration
	matrixProduct<<<blockSize,gridSize>>>(dev_Matrix1, dev_Matrix2, dev_Result);
	hipMemcpy(Result.elements, dev_Result.elements, MemSize, hipMemcpyDeviceToHost);
	
	gettimeofday(&end,NULL);
	printf("End Values %ld, %ld\n",end.tv_sec,end.tv_usec);
	printf("Matrix 1:\n");
	for(i=0;i<=MatSize;i++)
	{
		for(j=0;j<=MatSize;j++)
			printf("%d\t",Matrix3.elements[(i*Matrix3.width)+j]);
		printf("\n");
	}
	printf("Matrix 2:\n");
	for(i=0;i<=MatSize;i++)
		{
			for(j=0;j<=MatSize;j++)
				printf("%d\t",Matrix2.elements[(i*Matrix2.width)+j]);
			printf("\n");
		}
	printf("Result:\n");
	for(i=0;i<=MatSize;i++)
		{
			for(j=0;j<=MatSize;j++)
				printf("%d\t",Result.elements[i*Result.width+j]);
			printf("\n");
		}
	printf("Elapsed Time: %ld \n",/*((end.tv_sec-start.tv_sec)*1000000)+*/(end.tv_usec-start.tv_usec));
}

__global__ void matrixProduct(Matrix Mat1, Matrix Mat2, Matrix Res)
{
	int row = blockIdx.x;
	int col = blockIdx.y;
	int k,sum;
	sum=0;
	for(k=0;k<=Mat1.width;k++)
	{
		sum=sum+(Mat1.elements[(row*Mat1.width)+k])*(Mat2.elements[(k*Mat2.width)+col]);
	}
	Res.elements[(row*Res.width)+col]=sum;
}
