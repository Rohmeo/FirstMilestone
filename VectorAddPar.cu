
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void VecAdd(int *Vec1, int *Vec2, int *Res){
	Res[blockIdx.x]=Vec2[blockIdx.x]+Vec2[blockIdx.x];
}

int main(){
	int length, i;
	size_t size;
	int *dev_Vector1;
	int *dev_Vector2;
	int *dev_Result;
	
	length = 10;
	size = (length+1)*sizeof(int);
	
	int* Vector1 = (int*)malloc(size);
	int* Vector2 = (int*)malloc(size);
	int* Result  = (int*)malloc(size);
	
	for(i=0;i<=length;i++){
		Vector1[i] = i;
		Vector2[i] = i;
	}
	hipMalloc((void**)&dev_Vector1, size);
	hipMalloc((void**)&dev_Vector2, size);
	hipMalloc((void**)&dev_Result, size);
	
	hipMemcpy(dev_Vector1,Vector1,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_Vector2,Vector2,size,hipMemcpyHostToDevice);
	VecAdd<<<length+1,1>>>(dev_Vector1, dev_Vector2, dev_Result);
	
	hipMemcpy(Result, dev_Result, size, hipMemcpyDeviceToHost);
	
	for(i=0;i<=length;i++){
		printf("%d\t",Result[i]);
	}
	printf("\n");
	free(Vector1);
	free(Vector2);
	free(Result);
	hipFree(dev_Vector1);
	hipFree(dev_Vector2);
	hipFree(dev_Result);
	return 0;
}
